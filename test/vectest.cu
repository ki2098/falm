
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <vector>
#include <stdio.h>

using namespace std;

__global__ void vecaddone(double *vec, size_t size) {
    size_t id = blockDim.x*blockIdx.x + threadIdx.x;
    if (id < size) {
        vec[id] += 1;
    }
}

int main() {
    vector<double> x(100);
    for (size_t i = 0; i < x.size(); i ++) {
        x[i] = i;
    }
    double *ptr;
    hipMalloc(&ptr, sizeof(double)*x.size());
    hipMemcpy(ptr, x.data(), sizeof(double)*x.size(), hipMemcpyHostToDevice);
    vecaddone<<<8,8>>>(ptr, x.size());
    hipMemcpy(x.data(), ptr, sizeof(double)*x.size(), hipMemcpyDeviceToHost);
    for (size_t i = 0; i < x.size(); i ++) {
        printf("%lf\n", x[i]);
    }
}