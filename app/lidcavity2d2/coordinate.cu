#include "hip/hip_runtime.h"
#include "coordinate.h"
#include "../../src/util.h"
#include "../../src/dev/devutil.cuh"

namespace LidCavity2d2 {

using namespace Falm;

__global__ void kernel_setCoord(
    Real               side_lenth,
    Int                side_n_cell,
    Int3              pdm_shape,
    Int3              pdm_offset,
    Int                gc,
    const MatrixFrame<Real> *vx,
    const MatrixFrame<Real> *vh,
    const MatrixFrame<Real> *vkx,
    const MatrixFrame<Real> *vg,
    const MatrixFrame<Real> *vja
) {
    const MatrixFrame<Real> &x=*vx, &h=*vh, &kx=*vkx, &g=*vg, &ja=*vja;
    Int i, j, k;
    GLOBAL_THREAD_IDX_3D(i, j, k);
    if (i < pdm_shape[0] && j < pdm_shape[1] && k < pdm_shape[2]) {
        Int idx = IDX(i, j, k, pdm_shape);
        Real pitch = side_lenth / side_n_cell;
        Real dkdx = 1.0 / pitch;
        Real vol = pitch * pitch * pitch;
        x(idx, 0) = (i + pdm_offset[0] - gc + 0.5) * pitch;
        x(idx, 1) = (j + pdm_offset[1] - gc + 0.5) * pitch;
        x(idx, 2) = (k + pdm_offset[2] - gc      ) * pitch;
        h(idx, 0) = h(idx, 1) = h(idx, 2) = pitch;
        kx(idx, 0) = kx(idx, 1) = kx(idx, 2) = dkdx;
        ja(idx) = vol;
        g(idx, 0) = g(idx, 1) = g(idx, 2) = vol * dkdx * dkdx;
    }
}

void setCoord(
    Real          side_lenth,
    Int           side_n_cell,
    Region       &pdm,
    Int           gc,
    Matrix<Real> &x,
    Matrix<Real> &h,
    Matrix<Real> &kx,
    Matrix<Real> &g,
    Matrix<Real> &ja,
    dim3          block_dim
) {
    x.alloc(pdm.shape, 3, HDC::Device);
    h.alloc(pdm.shape, 3, HDC::Device);
    kx.alloc(pdm.shape, 3, HDC::Device);
    g.alloc(pdm.shape, 3, HDC::Device);
    ja.alloc(pdm.shape, 1, HDC::Device);

    dim3 grid_dim(
        (pdm.shape[0] + block_dim.x - 1) / block_dim.x,
        (pdm.shape[1] + block_dim.y - 1) / block_dim.y,
        (pdm.shape[2] + block_dim.z - 1) / block_dim.z
    );
    kernel_setCoord<<<grid_dim, block_dim, 0, 0>>>(
        side_lenth,
        side_n_cell,
        pdm.shape,
        pdm.offset,
        gc,
        x.devptr,
        h.devptr,
        kx.devptr,
        g.devptr,
        ja.devptr
    );
}

}