#include "hip/hip_runtime.h"
#include "boundaryCondition.h"
#include "../../src/util.h"
#include "../../src/dev/devutil.cuh"

namespace LidCavity2d {

using namespace Falm;

__global__ void kernel_pressureBC_E(
    MatrixFrame<REAL> &p,
    INTx3              pdm_shape
) {
    INT i, j, k;
    GLOBAL_THREAD_IDX_3D(i, j, k);
    if (i < 1 && j < pdm_shape.y - Gdx2 && k < pdm_shape.z - Gdx2) {
        i += pdm_shape.x - Gd;
        j += Gd;
        k += Gd;
        p(IDX(i, j, k, pdm_shape)) = p(IDX(i-1, j, k, pdm_shape));
    }
}

__global__ void kernel_pressureBC_W(
    MatrixFrame<REAL> &p,
    INTx3              pdm_shape
) {
    INT i, j, k;
    GLOBAL_THREAD_IDX_3D(i, j, k);
    if (i < 1 && j < pdm_shape.y - Gdx2 && k < pdm_shape.z - Gdx2) {
        i += Gd - 1;
        j += Gd;
        k += Gd;
        p(IDX(i, j, k, pdm_shape)) = p(IDX(i+1, j, k, pdm_shape));
    }
}

__global__ void kernel_pressureBC_N(
    MatrixFrame<REAL> &p,
    INTx3              pdm_shape
) {
    INT i, j, k;
    GLOBAL_THREAD_IDX_3D(i, j, k);
    if (i < pdm_shape.x - Gdx2 && j < 1 && k < pdm_shape.z - Gdx2) {
        i += Gd;
        j += pdm_shape.y - Gd;
        k += Gd;
        p(IDX(i, j, k, pdm_shape)) = p(IDX(i, j-1, k, pdm_shape));
    }
}

__global__ void kernel_pressureBC_S(
    MatrixFrame<REAL> &p,
    INTx3              pdm_shape
) {
    INT i, j, k;
    GLOBAL_THREAD_IDX_3D(i, j, k);
    if (i < pdm_shape.x - Gdx2 && j < 1 && k < pdm_shape.z - Gdx2) {
        i += Gd;
        j += Gd - 1;
        k += Gd;
        p(IDX(i, j, k, pdm_shape)) = p(IDX(i, j+1, k, pdm_shape));
    }
}

void pressureBC(
    Matrix<REAL> &p,
    Mapper       &pdm,
    STREAM       *streamptr
) {
    dim3 grid_dim_ew(1, (pdm.shape.y - Gdx2 + 7) / 8, (pdm.shape.z - Gdx2 + 7) / 8);
    dim3 block_dim_ew(1, 8, 8);
    STREAM stream_e = (streamptr)? streamptr[0] : (STREAM)0;
    kernel_pressureBC_E<<<grid_dim_ew, block_dim_ew, 0, stream_e>>>(*(p.devptr), pdm.shape);
    STREAM stream_w = (streamptr)? streamptr[1] : (STREAM)0;
    kernel_pressureBC_W<<<grid_dim_ew, block_dim_ew, 0, stream_w>>>(*(p.devptr), pdm.shape);

    dim3 grid_dim_ns((pdm.shape.x - Gdx2 + 7) / 8, 1, (pdm.shape.z - Gdx2 + 7) / 8);
    dim3 block_dim_ns(8, 1, 8);
    STREAM stream_n = (streamptr)? streamptr[2] : (STREAM)0;
    kernel_pressureBC_N<<<grid_dim_ns, block_dim_ns, 0, stream_n>>>(*(p.devptr), pdm.shape);
    STREAM stream_s = (streamptr)? streamptr[3] : (STREAM)0;
    kernel_pressureBC_S<<<grid_dim_ns, block_dim_ns, 0, stream_s>>>(*(p.devptr), pdm.shape);

    if (streamptr) {
        for (INT fid = 0; fid < 4; fid ++) {
            falmWaitStream(streamptr[fid]);
        }
    } else {
        falmWaitStream();
    }
}

__global__ void kernel_velocityBC_E(
    MatrixFrame<REAL> &u,
    INTx3              pdm_shape
) {
    INT i, j, k;
    GLOBAL_THREAD_IDX_3D(i, j, k);
    if (i < 1 && j < pdm_shape.y - Gdx2 && k < pdm_shape.z - Gdx2) {
        i += pdm_shape.x - Gd;
        j += Gd;
        k += Gd;
        REAL uboundary[] = {0.0, 0.0, 0.0};
        for (INT d = 0; d < 3; d ++) {
            u(IDX(i  , j, k, pdm_shape), d) = 2 * uboundary[d] - u(IDX(i-1, j, k, pdm_shape), d);
            u(IDX(i+1, j, k, pdm_shape), d) = 2 * uboundary[d] - u(IDX(i-2, j, k, pdm_shape), d);
        }
    }
}

__global__ void kernel_velocityBC_W(
    MatrixFrame<REAL> &u,
    INTx3              pdm_shape
) {
    INT i, j, k;
    GLOBAL_THREAD_IDX_3D(i, j, k);
    if (i < 1 && j < pdm_shape.y - Gdx2 && k < pdm_shape.z - Gdx2) {
        i += Gd - 1;
        j += Gd;
        k += Gd;
        REAL uboundary[] = {0.0, 0.0, 0.0};
        for (INT d = 0; d < 3; d ++) {
            u(IDX(i  , j, k, pdm_shape), d) = 2 * uboundary[d] - u(IDX(i+1, j, k, pdm_shape), d);
            u(IDX(i-1, j, k, pdm_shape), d) = 2 * uboundary[d] - u(IDX(i+2, j, k, pdm_shape), d);
        }
    }
}

__global__ void kernel_velocityBC_N(
    MatrixFrame<REAL> &u,
    INTx3              pdm_shape
) {
    INT i, j, k;
    GLOBAL_THREAD_IDX_3D(i, j, k);
    if (i < pdm_shape.x - Gdx2 && j < 1 && k < pdm_shape.z - Gdx2) {
        i += Gd;
        j += pdm_shape.y - Gd;
        k += Gd;
        REAL uboundary[] = {1.0, 0.0, 0.0};
        for (INT d = 0; d < 3; d ++) {
            u(IDX(i, j  , k, pdm_shape), d) = 2 * uboundary[d] - u(IDX(i, j-1, k, pdm_shape), d);
            u(IDX(i, j+1, k, pdm_shape), d) = 2 * uboundary[d] - u(IDX(i, j-2, k, pdm_shape), d);
        }
    }
}

__global__ void kernel_velocityBC_S(
    MatrixFrame<REAL> &u,
    INTx3              pdm_shape
) {
    INT i, j, k;
    GLOBAL_THREAD_IDX_3D(i, j, k);
    if (i < pdm_shape.x - Gdx2 && j < 1 && k < pdm_shape.z - Gdx2) {
        i += Gd;
        j += Gd - 1;
        k += Gd;
        REAL uboundary[] = {0.0, 0.0, 0.0};
        for (INT d = 0; d < 3; d ++) {
            u(IDX(i, j  , k, pdm_shape), d) = 2 * uboundary[d] - u(IDX(i, j+1, k, pdm_shape), d);
            u(IDX(i, j-1, k, pdm_shape), d) = 2 * uboundary[d] - u(IDX(i, j+2, k, pdm_shape), d);
        }
    }
}

void velocityBC(
    Matrix<REAL> &u,
    Mapper       &pdm,
    STREAM       *streamptr
) {
    dim3 grid_dim_ew(1, (pdm.shape.y - Gdx2 + 7) / 8, (pdm.shape.z - Gdx2 + 7) / 8);
    dim3 block_dim_ew(1, 8, 8);
    STREAM stream_e = (streamptr)? streamptr[0] : (STREAM)0;
    kernel_velocityBC_E<<<grid_dim_ew, block_dim_ew, 0, stream_e>>>(*(u.devptr), pdm.shape);
    STREAM stream_w = (streamptr)? streamptr[1] : (STREAM)0;
    kernel_velocityBC_W<<<grid_dim_ew, block_dim_ew, 0, stream_w>>>(*(u.devptr), pdm.shape);

    dim3 grid_dim_ns((pdm.shape.x - Gdx2 + 7) / 8, 1, (pdm.shape.z - Gdx2 + 7) / 8);
    dim3 block_dim_ns(8, 1, 8);
    STREAM stream_n = (streamptr)? streamptr[2] : (STREAM)0;
    kernel_velocityBC_N<<<grid_dim_ns, block_dim_ns, 0, stream_n>>>(*(u.devptr), pdm.shape);
    STREAM stream_s = (streamptr)? streamptr[3] : (STREAM)0;
    kernel_velocityBC_S<<<grid_dim_ns, block_dim_ns, 0, stream_s>>>(*(u.devptr), pdm.shape);

    if (streamptr) {
        for (INT fid = 0; fid < 4; fid ++) {
            falmWaitStream(streamptr[fid]);
        }
    } else {
        falmWaitStream();
    }
}

__global__ void kernel_forceFaceVelocityZero_E(
    MatrixFrame<REAL> &uu,
    INTx3              pdm_shape
) {
    INT i, j, k;
    GLOBAL_THREAD_IDX_3D(i, j, k);
    if (i < 1 && j < pdm_shape.y - Gdx2 && k < pdm_shape.z - Gdx2) {
        i += pdm_shape.x - Gd - 1;
        j += Gd;
        k += Gd;
        uu(IDX(i, j, k, pdm_shape), 0) = 0;
    }
}

__global__ void kernel_forceFaceVelocityZero_W(
    MatrixFrame<REAL> &uu,
    INTx3              pdm_shape
) {
    INT i, j, k;
    GLOBAL_THREAD_IDX_3D(i, j, k);
    if (i < 1 && j < pdm_shape.y - Gdx2 && k < pdm_shape.z - Gdx2) {
        i += Gd - 1;
        j += Gd;
        k += Gd;
        uu(IDX(i, j, k, pdm_shape), 0) = 0;
    }
}

__global__ void kernel_forceFaceVelocityZero_N(
    MatrixFrame<REAL> &uu,
    INTx3              pdm_shape
) {
    INT i, j, k;
    GLOBAL_THREAD_IDX_3D(i, j, k);
    if (i < pdm_shape.x - Gdx2 && j < 1 && k < pdm_shape.z - Gdx2) {
        i += Gd;
        j += pdm_shape.y - Gd - 1;
        k += Gd;
        uu(IDX(i, j, k, pdm_shape), 1) = 0;
    }
}

__global__ void kernel_forceFaceVelocityZero_S(
    MatrixFrame<REAL> &uu,
    INTx3              pdm_shape
) {
    INT i, j, k;
    GLOBAL_THREAD_IDX_3D(i, j, k);
    if (i < pdm_shape.x - Gdx2 && j < 1 && k < pdm_shape.z - Gdx2) {
        i += Gd;
        j += Gd - 1;
        k += Gd;
        uu(IDX(i, j, k, pdm_shape), 1) = 0;
    }
}

void forceFaceVelocityZero(
    Matrix<REAL> &uu,
    Mapper       &pdm,
    STREAM       *streamptr
) {
    dim3 grid_dim_ew(1, (pdm.shape.y - Gdx2 + 7) / 8, (pdm.shape.z - Gdx2 + 7) / 8);
    dim3 block_dim_ew(1, 8, 8);
    STREAM stream_e = (streamptr)? streamptr[0] : (STREAM)0;
    kernel_forceFaceVelocityZero_E<<<grid_dim_ew, block_dim_ew, 0, stream_e>>>(*(uu.devptr), pdm.shape);
    STREAM stream_w = (streamptr)? streamptr[1] : (STREAM)0;
    kernel_forceFaceVelocityZero_W<<<grid_dim_ew, block_dim_ew, 0, stream_w>>>(*(uu.devptr), pdm.shape);

    dim3 grid_dim_ns((pdm.shape.x - Gdx2 + 7) / 8, 1, (pdm.shape.z - Gdx2 + 7) / 8);
    dim3 block_dim_ns(8, 1, 8);
    STREAM stream_n = (streamptr)? streamptr[2] : (STREAM)0;
    kernel_forceFaceVelocityZero_N<<<grid_dim_ns, block_dim_ns, 0, stream_n>>>(*(uu.devptr), pdm.shape);
    STREAM stream_s = (streamptr)? streamptr[3] : (STREAM)0;
    kernel_forceFaceVelocityZero_S<<<grid_dim_ns, block_dim_ns, 0, stream_s>>>(*(uu.devptr), pdm.shape);

    if (streamptr) {
        for (INT fid = 0; fid < 4; fid ++) {
            falmWaitStream(streamptr[fid]);
        }
    } else {
        falmWaitStream();
    }
}

void copyZ5(
    Matrix<REAL> &field,
    Mapper       &pdm,
    STREAM       *streamptr
) {
    INT idxcc = IDX(0, 0, Gd  , pdm.shape);
    INT idxt1 = IDX(0, 0, Gd+1, pdm.shape);
    INT idxt2 = IDX(0, 0, Gd+2, pdm.shape);
    INT idxb1 = IDX(0, 0, Gd-1, pdm.shape);
    INT idxb2 = IDX(0, 0, Gd-2, pdm.shape);
    INT slice_size = pdm.shape.x * pdm.shape.y;
    for (INT d = 0; d < field.shape.y; d ++) {
        falmMemcpyAsync(&field.dev(idxt1, d), &field.dev(idxcc, d), sizeof(REAL) * slice_size, MCpType::Dev2Dev);
        falmMemcpyAsync(&field.dev(idxt2, d), &field.dev(idxcc, d), sizeof(REAL) * slice_size, MCpType::Dev2Dev);
        falmMemcpyAsync(&field.dev(idxb1, d), &field.dev(idxcc, d), sizeof(REAL) * slice_size, MCpType::Dev2Dev);
        falmMemcpyAsync(&field.dev(idxb2, d), &field.dev(idxcc, d), sizeof(REAL) * slice_size, MCpType::Dev2Dev);
    }
    falmWaitStream();
}

}