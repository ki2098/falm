#include "hip/hip_runtime.h"
#include "../typedef.h"
#include "../CPMDev.h"
#include "devutil.cuh"

namespace Falm {

__global__ void kernel_CPM_PackBuffer(double *buffer, uint3 buf_shape, uint3 buf_offset, double *src, uint3 src_shape) {
    unsigned int i, j, k;
    GLOBAL_THREAD_IDX_3D(i, j, k);
    if (i < buf_shape.x && j < buf_shape.y && k < buf_shape.z) {
        unsigned int buf_idx = IDX(i, j, k, buf_shape);
        i += buf_offset.x;
        j += buf_offset.y;
        k += buf_offset.z;
        unsigned int src_idx = IDX(i, j, k, src_shape);
        buffer[buf_idx] = src[src_idx];
    }
}

void dev_CPM_PackBuffer(CPMBuffer<double> &buffer, double *src, Mapper &pdom, dim3 &block_dim) {
    Mapper &map = buffer.map;
    dim3 grid_dim(
        (map.shape.x + block_dim.x - 1) / block_dim.x,
        (map.shape.y + block_dim.y - 1) / block_dim.y,
        (map.shape.z + block_dim.z - 1) / block_dim.z
    );
    kernel_CPM_PackBuffer<<<grid_dim, block_dim, 0, 0>>>(buffer.ptr, buffer.map.shape, buffer.map.offset, src, pdom.shape);
}

__global__ void kernel_CPM_PackColoredBuffer(double *buffer, uint3 buf_shape, uint3 buf_offset, unsigned int color, double *src, uint3 src_shape, uint3 src_offset) {
    unsigned int i, j, k;
    GLOBAL_THREAD_IDX_3D(i, j, k);
    if (i < buf_shape.x && j < buf_shape.y && k < buf_shape.z) {
        unsigned int buf_idx = IDX(i, j, k, buf_shape);
        i += buf_offset.x;
        j += buf_offset.y;
        k += buf_offset.z;
        unsigned int src_idx = IDX(i, j, k, src_shape);
        if ((i + j + k + SUM3(src_offset)) % 2 == color) {
            buffer[buf_idx / 2] = src[src_idx];
        }
    }
}

void dev_CPM_PackColoredBuffer(CPMBuffer<double> &buffer, double *src, Mapper &pdom, dim3 &block_dim) {
    Mapper &map = buffer.map;
    dim3 grid_dim(
        (map.shape.x + block_dim.x - 1) / block_dim.x,
        (map.shape.y + block_dim.y - 1) / block_dim.y,
        (map.shape.z + block_dim.z - 1) / block_dim.z
    );
    kernel_CPM_PackColoredBuffer<<<grid_dim, block_dim, 0, 0>>>(buffer.ptr, buffer.map.shape, buffer.map.offset, buffer.color, src, pdom.shape, pdom.offset);
}

__global__ void kernel_CPM_UnpackBuffer(double *buffer, uint3 buf_shape, uint3 buf_offset, double *dst, uint3 dst_shape) {
    unsigned int i, j, k;
    GLOBAL_THREAD_IDX_3D(i, j, k);
    if (i < buf_shape.x && j < buf_shape.y && k < buf_shape.z) {
        unsigned int buf_idx = IDX(i, j, k, buf_shape);
        i += buf_offset.x;
        j += buf_offset.y;
        k += buf_offset.z;
        unsigned int dst_idx = IDX(i, j, k, dst_shape);
        dst[dst_idx] = buffer[buf_idx];
    }
}

void dev_CPM_UnpackBuffer(CPMBuffer<double> &buffer, double *dst, Mapper &pdom, dim3 &block_dim) {
    Mapper &map = buffer.map;
    dim3 grid_dim(
        (map.shape.x + block_dim.x - 1) / block_dim.x,
        (map.shape.y + block_dim.y - 1) / block_dim.y,
        (map.shape.z + block_dim.z - 1) / block_dim.z
    );
    kernel_CPM_UnpackBuffer<<<grid_dim, block_dim, 0, 0>>>(buffer.ptr, buffer.map.shape, buffer.map.offset, dst, pdom.shape);
}

__global__ void kernel_CPM_UnpackColoredBuffer(double *buffer, uint3 buf_shape, uint3 buf_offset, unsigned int color , double *dst, uint3 dst_shape, uint3 dst_offset) {
    unsigned int i, j, k;
    GLOBAL_THREAD_IDX_3D(i, j, k);
    if (i < buf_shape.x && j < buf_shape.y && k < buf_shape.z) {
        unsigned int buf_idx = IDX(i, j, k, buf_shape);
        i += buf_offset.x;
        j += buf_offset.y;
        k += buf_offset.z;
        unsigned int dst_idx = IDX(i, j, k, dst_shape);
        if ((i + j + k + SUM3(dst_offset)) % 2 == color) {
            dst[dst_idx] = buffer[buf_idx / 2];
        }
    }
}

void dev_CPM_UnpackColoredBuffer(CPMBuffer<double> &buffer, double *dst, Mapper &pdom, dim3 &block_dim) {
    Mapper &map = buffer.map;
    dim3 grid_dim(
        (map.shape.x + block_dim.x - 1) / block_dim.x,
        (map.shape.y + block_dim.y - 1) / block_dim.y,
        (map.shape.z + block_dim.z - 1) / block_dim.z
    );
    kernel_CPM_UnpackColoredBuffer<<<grid_dim, block_dim, 0, 0>>>(buffer.ptr, buffer.map.shape, buffer.map.offset, buffer.color, dst, pdom.shape, pdom.offset);
}

}