#include "hip/hip_runtime.h"
#include "../typedef.h"
#include "../CPML1.h"
#include "devutil.cuh"

namespace Falm {

__global__ void kernel_CPM_PackBuffer(double *buffer, uint3 buf_shape, uint3 buf_offset, double *src, uint3 src_shape) {
    unsigned int i, j, k;
    GLOBAL_THREAD_IDX_3D(i, j, k);
    if (i < buf_shape.x && j < buf_shape.y && k < buf_shape.z) {
        unsigned int buf_idx = IDX(i, j, k, buf_shape);
        i += buf_offset.x;
        j += buf_offset.y;
        k += buf_offset.z;
        unsigned int src_idx = IDX(i, j, k, src_shape);
        buffer[buf_idx] = src[src_idx];
    }
}

void CPML1dev_PackBuffer(CPMBuffer<double> &buffer, double *src, Mapper &pdm, dim3 block_dim) {
    Mapper &map = buffer.map;
    dim3 grid_dim(
        (map.shape.x + block_dim.x - 1) / block_dim.x,
        (map.shape.y + block_dim.y - 1) / block_dim.y,
        (map.shape.z + block_dim.z - 1) / block_dim.z
    );
    if (buffer.hdctype == HDCType::Device) {
        kernel_CPM_PackBuffer<<<grid_dim, block_dim, 0, 0>>>(buffer.ptr, buffer.map.shape, buffer.map.offset, src, pdm.shape);
    } else if (buffer.hdctype == HDCType::Host) {
        double *ptr = (double*)falmDevMalloc(sizeof(double) * buffer.size);
        kernel_CPM_PackBuffer<<<grid_dim, block_dim, 0, 0>>>(ptr, buffer.map.shape, buffer.map.offset, src, pdm.shape);
        falmMemcpy(buffer.ptr, ptr, sizeof(double) * buffer.size, MCpType::Dev2Hst);
        falmDevFreePtr(ptr);
    }
}

__global__ void kernel_CPM_PackColoredBuffer(double *buffer, uint3 buf_shape, uint3 buf_offset, unsigned int color, double *src, uint3 src_shape, uint3 src_offset) {
    unsigned int i, j, k;
    GLOBAL_THREAD_IDX_3D(i, j, k);
    if (i < buf_shape.x && j < buf_shape.y && k < buf_shape.z) {
        unsigned int buf_idx = IDX(i, j, k, buf_shape);
        i += buf_offset.x;
        j += buf_offset.y;
        k += buf_offset.z;
        unsigned int src_idx = IDX(i, j, k, src_shape);
        if ((i + j + k + SUM3(src_offset)) % 2 == color) {
            buffer[buf_idx / 2] = src[src_idx];
        }
    }
}

void CPML1dev_PackColoredBuffer(CPMBuffer<double> &buffer, double *src, Mapper &pdm, dim3 block_dim) {
    Mapper &map = buffer.map;
    dim3 grid_dim(
        (map.shape.x + block_dim.x - 1) / block_dim.x,
        (map.shape.y + block_dim.y - 1) / block_dim.y,
        (map.shape.z + block_dim.z - 1) / block_dim.z
    );
    if (buffer.hdctype == HDCType::Device) {
        kernel_CPM_PackColoredBuffer<<<grid_dim, block_dim, 0, 0>>>(buffer.ptr, buffer.map.shape, buffer.map.offset, buffer.color, src, pdm.shape, pdm.offset);
    } else if (buffer.hdctype == HDCType::Host) {
        double *ptr = (double*)falmDevMalloc(sizeof(double) * buffer.size);
        kernel_CPM_PackColoredBuffer<<<grid_dim, block_dim, 0, 0>>>(ptr, buffer.map.shape, buffer.map.offset, buffer.color, src, pdm.shape, pdm.offset);
        falmMemcpy(buffer.ptr, ptr, sizeof(double) * buffer.size, MCpType::Dev2Hst);
        falmDevFreePtr(ptr);
    }
}

__global__ void kernel_CPM_UnpackBuffer(double *buffer, uint3 buf_shape, uint3 buf_offset, double *dst, uint3 dst_shape) {
    unsigned int i, j, k;
    GLOBAL_THREAD_IDX_3D(i, j, k);
    if (i < buf_shape.x && j < buf_shape.y && k < buf_shape.z) {
        unsigned int buf_idx = IDX(i, j, k, buf_shape);
        i += buf_offset.x;
        j += buf_offset.y;
        k += buf_offset.z;
        unsigned int dst_idx = IDX(i, j, k, dst_shape);
        dst[dst_idx] = buffer[buf_idx];
    }
}

void CPML1dev_UnpackBuffer(CPMBuffer<double> &buffer, double *dst, Mapper &pdm, dim3 block_dim) {
    Mapper &map = buffer.map;
    dim3 grid_dim(
        (map.shape.x + block_dim.x - 1) / block_dim.x,
        (map.shape.y + block_dim.y - 1) / block_dim.y,
        (map.shape.z + block_dim.z - 1) / block_dim.z
    );
    if (buffer.hdctype == HDCType::Device) {
        kernel_CPM_UnpackBuffer<<<grid_dim, block_dim, 0, 0>>>(buffer.ptr, buffer.map.shape, buffer.map.offset, dst, pdm.shape);
    } else if (buffer.hdctype == HDCType::Host) {
        double *ptr = (double*)falmDevMalloc(sizeof(double) * buffer.size);
        falmMemcpy(ptr, buffer.ptr, sizeof(double) * buffer.size, MCpType::Hst2Dev);
        kernel_CPM_UnpackBuffer<<<grid_dim, block_dim, 0, 0>>>(ptr, buffer.map.shape, buffer.map.offset, dst, pdm.shape);
        falmDevFreePtr(ptr);
    }
    
}

__global__ void kernel_CPM_UnpackColoredBuffer(double *buffer, uint3 buf_shape, uint3 buf_offset, unsigned int color , double *dst, uint3 dst_shape, uint3 dst_offset) {
    unsigned int i, j, k;
    GLOBAL_THREAD_IDX_3D(i, j, k);
    if (i < buf_shape.x && j < buf_shape.y && k < buf_shape.z) {
        unsigned int buf_idx = IDX(i, j, k, buf_shape);
        i += buf_offset.x;
        j += buf_offset.y;
        k += buf_offset.z;
        unsigned int dst_idx = IDX(i, j, k, dst_shape);
        if ((i + j + k + SUM3(dst_offset)) % 2 == color) {
            dst[dst_idx] = buffer[buf_idx / 2];
        }
    }
}

void CPML1dev_UnpackColoredBuffer(CPMBuffer<double> &buffer, double *dst, Mapper &pdm, dim3 block_dim) {
    Mapper &map = buffer.map;
    dim3 grid_dim(
        (map.shape.x + block_dim.x - 1) / block_dim.x,
        (map.shape.y + block_dim.y - 1) / block_dim.y,
        (map.shape.z + block_dim.z - 1) / block_dim.z
    );
    if (buffer.hdctype == HDCType::Device) {
        kernel_CPM_UnpackColoredBuffer<<<grid_dim, block_dim, 0, 0>>>(buffer.ptr, buffer.map.shape, buffer.map.offset, buffer.color, dst, pdm.shape, pdm.offset);
    } else if (buffer.hdctype == HDCType::Host) {
        double *ptr = (double*)falmDevMalloc(sizeof(double) * buffer.size);
        falmMemcpy(ptr, buffer.ptr, sizeof(double) * buffer.size, MCpType::Hst2Dev);
        kernel_CPM_UnpackColoredBuffer<<<grid_dim, block_dim, 0, 0>>>(ptr, buffer.map.shape, buffer.map.offset, buffer.color, dst, pdm.shape, pdm.offset);
        falmDevFreePtr(ptr);
    }
}

}
