#include "hip/hip_runtime.h"
#include <math.h>
#include "almDevCall.h"
#include "../dev/devutil.cuh"
#include "../falmath.h"
#include <thrust/device_ptr.h>
#include <thrust/reduce.h>

namespace Falm {

namespace Alm {

template <typename T>
__host__ __device__ static Int find_index(T *seq, T value, Int size) {
    if (value < seq[0]) {
        return -1;
    } else if (value >= seq[size-1]) {
        return size-1;
    } else {
        for (Int i = 0; i < size-1; i ++) {
            if (seq[i] <= value && value < seq[i+1]) {
                return i;
            }
        }
        return size-1;
    }
}

__host__ __device__ inline Real trilinear_interpolation(
    Real xd, Real yd, Real zd,
    Real c0, Real c1, Real c2, Real c3, Real c4, Real c5, Real c6, Real c7
) {
    Real c01 = c0*(1. - xd) + c1*xd;
    Real c23 = c2*(1. - xd) + c3*xd;
    Real c45 = c4*(1. - xd) + c5*xd;
    Real c67 = c6*(1. - xd) + c7*xd;

    Real c0123 = c01*(1. - yd) + c23*yd;
    Real c4567 = c45*(1. - yd) + c67*yd;

    return c0123*(1. - zd) + c4567*zd;
}

__global__ void kernel_UpdateTurbineAngles(
    TurbineFrame *vturbines,
    Real t
) {
    TurbineFrame &turbines = *vturbines;
    size_t thread_id = GLOBAL_THREAD_IDX();
    if (thread_id < turbines.n_turbine) {
        auto angle_type = int(turbines.angle_type[thread_id]);
        auto motion = turbines.motion[thread_id];
        Real3 angle{{0,0,0}}, angular_velocity{{0,0,0}};
        if (angle_type) {
            angle[angle_type - 1] = motion[0]*sin(motion[1]*t + motion[2]);
            angular_velocity[angle_type - 1] = motion[0]*motion[1]*cos(motion[1]*t + motion[2]);
        }
        turbines.angle[thread_id] = angle;
        turbines.angular_velocity[thread_id] = angular_velocity;
        // printf("turbine %d (%lf %lf %lf) (%lf %lf %lf)\n", thread_id, turbines.angle[thread_id][0], turbines.angle[thread_id][1], turbines.angle[thread_id][2], turbines.angular_velocity[thread_id][0], turbines.angular_velocity[thread_id][1], turbines.angular_velocity[thread_id][2]);
    }
}

__global__ void kernel_UpdateAPX(
    MatrixFrame<Real> *vx,
    MatrixFrame<Real> *vy,
    MatrixFrame<Real> *vz,
    // MatrixFrame<INT> *vxoffset,
    // MatrixFrame<INT> *vyoffset,
    // MatrixFrame<INT> *vzoffset,
    TurbineFrame *vturbines,
    APFrame *vaps,
    Int n_ap_per_blade,
    Real t,
    Int gc,
    int rank
) {
    const MatrixFrame<Real> &x = *vx;
    const MatrixFrame<Real> &y = *vy;
    const MatrixFrame<Real> &z = *vz;
    // const MatrixFrame<INT> &xoffset = *vxoffset;
    // const MatrixFrame<INT> &yoffset = *vyoffset;
    // const MatrixFrame<INT> &zoffset = *vzoffset;
    const TurbineFrame &turbines = *vturbines;
    APFrame &aps = *vaps;

    size_t thread_id = GLOBAL_THREAD_IDX();
    if (thread_id < aps.apcount) {
        const Int ap_id = thread_id;
        const Int n_turbine = turbines.n_turbine;
        const Int n_blade = turbines.n_blade;
        const Int n_ap_per_turbine = n_ap_per_blade*n_blade;
        const Int turbine_id = ap_id/n_ap_per_turbine;
        const Int blade_id = (ap_id%n_ap_per_turbine)/n_ap_per_blade;
        const Real tip = turbines.tip_rate[turbine_id];

        double theta0 = (2*Pi/n_blade)*blade_id;
        // t = floormod(t, 2*Pi/tip);
        double theta  = tip*t + theta0;

        const Real3 &hub = turbines.hub[turbine_id];
        const Real3 &base = turbines.base[turbine_id];
        const Real3 &angle = turbines.angle[turbine_id];
        const EulerAngle angle_type = turbines.angle_type[turbine_id];
        const Real apr = aps.r[ap_id];

        Real3 coordinate1 = hub;
        coordinate1[1] += apr*cos(theta);
        coordinate1[2] += apr*sin(theta);
        Real3 coordinate0 = one_angle_frame_rotation(coordinate1, - angle, angle_type) + base;

        aps.xyz[ap_id] = coordinate0;
        Int apI = find_index(x.ptr, coordinate0[0], x.size);
        Int apJ = find_index(y.ptr, coordinate0[1], y.size);
        Int apK = find_index(z.ptr, coordinate0[2], z.size);
        aps.ijk[ap_id] = Int3{{apI, apJ, apK}};

        if (apI >= gc-1 && apI < x.size-gc-1 && apJ >= gc-1 && apJ < y.size-gc-1 && apK >= gc-1 && apK < z.size-gc-1) {
            aps.rank[ap_id] = rank;
        } else {
            aps.rank[ap_id] = -1;
        }

        // INT apRankI = find_index(xoffset.ptr, apI, xoffset.size);
        // INT apRankJ = find_index(yoffset.ptr, apJ, yoffset.size);
        // INT apRankK = find_index(zoffset.ptr, apK, zoffset.size);
        // aps.rank[ap_id] = IDX(apRankI, apRankJ, apRankK, mpi_shape);
        // printf("%d %d: %d (%lf %lf %lf) (%d %d %d)\n", rank, ap_id, aps.rank[ap_id], coordinate0[0], coordinate0[1], coordinate0[2], apI, apJ, apK);
    }
}

__global__ void kernel_CalcAPForce(
    MatrixFrame<Real> *vx,
    MatrixFrame<Real> *vy,
    MatrixFrame<Real> *vz,
    MatrixFrame<Real> *vuvw,
    TurbineFrame *vturbines,
    APFrame *vaps,
    Int n_ap_per_blade,
    Int3 shape,
    Real t,
    int rank
) {
    const MatrixFrame<Real> &x = *vx;
    const MatrixFrame<Real> &y = *vy;
    const MatrixFrame<Real> &z = *vz;
    const MatrixFrame<Real> &uvw = *vuvw;
    const TurbineFrame &turbines = *vturbines;
    APFrame &aps = *vaps;

    size_t thread_id = GLOBAL_THREAD_IDX();
    if (thread_id < aps.apcount) {
        const Int ap_id = thread_id;
        const Int n_turbine = turbines.n_turbine;
        const Int n_blade = turbines.n_blade;
        const Int n_ap_per_turbine = n_ap_per_blade*n_blade;
        const Int turbine_id = ap_id/n_ap_per_turbine;
        const Int blade_id = (ap_id%n_ap_per_turbine)/n_ap_per_blade;
        const Real tip = turbines.tip_rate[turbine_id];
        const Real3 &hub = turbines.hub[turbine_id];
        const Real3 &angle = turbines.angle[turbine_id];
        const Real3 &angular_velocity = turbines.angular_velocity[turbine_id];
        const EulerAngle angle_type = turbines.angle_type[turbine_id];
        const Real apr = aps.r[ap_id];
        const Real dr_per_ap = (turbines.radius - turbines.hub_radius)/n_ap_per_blade;
        if (aps.rank[ap_id] == rank) {
            // printf("%lf\n", dr_per_ap);
            Int3 apijk = aps.ijk[ap_id];
            Int  i0 = apijk[0], i1 = apijk[0] + 1;
            Int  j0 = apijk[1], j1 = apijk[1] + 1;
            Int  k0 = apijk[2], k1 = apijk[2] + 1;
            Real x0 = x(i0), x1 = x(i1);
            Real y0 = y(j0), y1 = y(j1);
            Real z0 = z(k0), z1 = z(k1);
            Real3 apxyz = aps.xyz[ap_id];
            Real xd = fabs(apxyz[0] - x0)/fabs(x1 - x0);
            Real yd = fabs(apxyz[1] - y0)/fabs(y1 - y0);
            Real zd = fabs(apxyz[2] - z0)/fabs(z1 - z0);
            Int idx0 = IDX(i0, j0, k0, shape);
            Int idx1 = IDX(i1, j0, k0, shape);
            Int idx2 = IDX(i0, j1, k0, shape);
            Int idx3 = IDX(i1, j1, k0, shape);
            Int idx4 = IDX(i0, j0, k1, shape);
            Int idx5 = IDX(i1, j0, k1, shape);
            Int idx6 = IDX(i0, j1, k1, shape);
            Int idx7 = IDX(i1, j1, k1, shape);

            Real c0, c1, c2, c3, c4, c5, c6, c7;
            const Int dim_u = 0;
            c0 = uvw(idx0, dim_u);
            c1 = uvw(idx1, dim_u);
            c2 = uvw(idx2, dim_u);
            c3 = uvw(idx3, dim_u);
            c4 = uvw(idx4, dim_u);
            c5 = uvw(idx5, dim_u);
            c6 = uvw(idx6, dim_u);
            c7 = uvw(idx7, dim_u);
            Real u_at_ap = trilinear_interpolation(xd, yd, zd, c0, c1, c2, c3, c4, c5, c6, c7);
            const Int dim_v = 1;
            c0 = uvw(idx0, dim_v);
            c1 = uvw(idx1, dim_v);
            c2 = uvw(idx2, dim_v);
            c3 = uvw(idx3, dim_v);
            c4 = uvw(idx4, dim_v);
            c5 = uvw(idx5, dim_v);
            c6 = uvw(idx6, dim_v);
            c7 = uvw(idx7, dim_v);
            Real v_at_ap = trilinear_interpolation(xd, yd, zd, c0, c1, c2, c3, c4, c5, c6, c7);
            const Int dim_w = 2;
            c0 = uvw(idx0, dim_w);
            c1 = uvw(idx1, dim_w);
            c2 = uvw(idx2, dim_w);
            c3 = uvw(idx3, dim_w);
            c4 = uvw(idx4, dim_w);
            c5 = uvw(idx5, dim_w);
            c6 = uvw(idx6, dim_w);
            c7 = uvw(idx7, dim_w);
            Real w_at_ap = trilinear_interpolation(xd, yd, zd, c0, c1, c2, c3, c4, c5, c6, c7);

            Real3 base = turbines.base[turbine_id];
            Real3 base_velocity = turbines.base_velocity[turbine_id];
            Real3 apxyz_tt = one_angle_frame_rotation(apxyz - base, angle, angle_type);
            Real3 uvw_at_ap_tt = one_angle_frame_rotation_dt(apxyz - base, Real3{{u_at_ap, v_at_ap, w_at_ap}} - base_velocity, angle, angular_velocity, angle_type);

            Real theta0 = (2*Pi/n_blade)*blade_id;
            // t = floormod(t, 2*Pi/tip);
            Real theta  = tip*t + theta0;

            Real ux_tt = uvw_at_ap_tt[0];
            Real ut_tt = tip*apr + uvw_at_ap_tt[1]*sin(theta) - uvw_at_ap_tt[2]*cos(theta);
            Real urel2 = ux_tt*ux_tt + ut_tt*ut_tt;
            Real phi = atan(ux_tt/ut_tt);
            Real chord, twist, cl, cd;
            aps.get_airfoil_params(ap_id, rad2deg(phi), chord, twist, cl, cd);

            // printf("%lf\n", dr_per_ap);
            Real fl = .5*cl*urel2*chord*dr_per_ap;
            Real fd = .5*cd*urel2*chord*dr_per_ap;
            Real fx = fl*cos(phi) + fd*sin(phi);
            Real ft = fl*sin(phi) - fd*sin(phi);
            ft *= sign(tip);
            Real3 ff_tt{{-fx, ft*sin(theta), -ft*cos(theta)}};
            aps.force[ap_id] = one_angle_frame_rotation(ff_tt, - angle, angle_type);
            aps.torque[ap_id] = fabs(ft)*apr;
            aps.thrust[ap_id] = fx;
        } else {
            aps.force[ap_id][0] = 0;
            aps.force[ap_id][1] = 0;
            aps.force[ap_id][2] = 0;
            aps.torque[ap_id] = 0;
            aps.thrust[ap_id] = 0;
        }
    }
}

__global__ void kernel_DistributeAPForce(
    MatrixFrame<Real> *vx,
    MatrixFrame<Real> *vy,
    MatrixFrame<Real> *vz,
    MatrixFrame<Real> *vff,
    APFrame *vaps,
    Real euler_eps,
    Int3 shape,
    Int gc
) {
    const MatrixFrame<Real> &x = *vx;
    const MatrixFrame<Real> &y = *vy;
    const MatrixFrame<Real> &z = *vz;
    MatrixFrame<Real> &ff = *vff;
    APFrame &aps = *vaps;
    Int i, j, k;
    GLOBAL_THREAD_IDX_3D(i, j, k);
    if (i < shape[0]-2*gc && j < shape[1]-2*gc && k < shape[2]-2*gc) {
        i += gc; j += gc; k += gc;
        Real cx = x(i);
        Real cy = y(j);
        Real cz = z(k);
        Real ffx = 0.;
        Real ffy = 0.;
        Real ffz = 0.;
        Real eta = 1./cubic(euler_eps*sqrt(Pi));
        for (Int ap_id = 0; ap_id < aps.apcount; ap_id ++) {
            const Real3 apxyz = aps.xyz[ap_id];
            Real rr2 = square(cx - apxyz[0]) + square(cy - apxyz[1]) + square(cz - apxyz[2]);

            ffx += aps.force[ap_id][0]*eta*exp(-rr2/square(euler_eps));
            ffy += aps.force[ap_id][1]*eta*exp(-rr2/square(euler_eps));
            ffz += aps.force[ap_id][2]*eta*exp(-rr2/square(euler_eps));
        }
        Int cid = IDX(i, j, k, shape);
        ff(cid, 0) = ffx;
        ff(cid, 1) = ffy;
        ff(cid, 2) = ffz;
    }
}

__global__ void kernel_DryDistribution(
    MatrixFrame<Real> *vx,
    MatrixFrame<Real> *vy,
    MatrixFrame<Real> *vz,
    MatrixFrame<Real> *vphi,
    APFrame *vaps,
    Real euler_eps,
    Int3 shape,
    Int gc
) {
    const MatrixFrame<Real> &x = *vx;
    const MatrixFrame<Real> &y = *vy;
    const MatrixFrame<Real> &z = *vz;
    MatrixFrame<Real> &phi = *vphi;
    APFrame &aps = *vaps;
    Int i, j, k;
    GLOBAL_THREAD_IDX_3D(i, j, k);
    if (i < shape[0]-2*gc && j < shape[1]-2*gc && k < shape[2]-2*gc) {
        i += gc; j += gc; k += gc;
        Real cx = x(i);
        Real cy = y(j);
        Real cz = z(k);
        Real cphi = 0;
        Real eta = 1./cubic(euler_eps*sqrt(Pi));
        for (Int ap_id = 0; ap_id < aps.apcount; ap_id ++) {
            const Real3 apxyz = aps.xyz[ap_id];
            Real rr2 = square(cx - apxyz[0]) + square(cy - apxyz[1]) + square(cz - apxyz[2]);
            cphi += eta*exp(-rr2/square(euler_eps));
        }
        phi(IDX(i,j,k,shape)) = cphi;
    }
}

void AlmDevCall::UpdateTurbineAngles(Real t, size_t block_size) {
    size_t block_number = (turbines.n_turbine + block_size - 1)/block_size;
    kernel_UpdateTurbineAngles<<<block_number, block_size>>>(turbines.devptr, t);
}

void AlmDevCall::UpdateAPX(Matrix<Real> &x, Matrix<Real> &y, Matrix<Real> &z, Real t, size_t block_size) {
    size_t block_number = (aps.apcount + block_size - 1)/block_size;
    kernel_UpdateAPX<<<block_number, block_size>>>(x.devptr, y.devptr, z.devptr, turbines.devptr, aps.devptr, n_ap_per_blade, t, gc, rank);
}

void AlmDevCall::CalcAPForce(Matrix<Real> &x, Matrix<Real> &y, Matrix<Real> &z, Matrix<Real> &uvw, Real t, size_t block_size) {
    size_t block_number = (aps.apcount + block_size - 1)/block_size;
    kernel_CalcAPForce<<<block_number, block_size>>>(x.devptr, y.devptr, z.devptr, uvw.devptr, turbines.devptr, aps.devptr, n_ap_per_blade, pdm_shape, t, rank);
}

void AlmDevCall::DistributeAPForce(Matrix<Real> &x, Matrix<Real> &y, Matrix<Real> &z, Matrix<Real> &ff, Real euler_eps, dim3 block_size) {
    dim3 block_number(
        (pdm_shape[0] + block_size.x - 1) / block_size.x,
        (pdm_shape[1] + block_size.y - 1) / block_size.y,
        (pdm_shape[2] + block_size.z - 1) / block_size.z
    );
    kernel_DistributeAPForce<<<block_number, block_size>>>(x.devptr, y.devptr, z.devptr, ff.devptr, aps.devptr, euler_eps, pdm_shape, gc);
}

void AlmDevCall::DryDistribution(Matrix<Real> &x, Matrix<Real> &y, Matrix<Real> &z, Matrix<Real> &phi, Real euler_eps, dim3 block_size) {
    dim3 block_number(
        (pdm_shape[0] + block_size.x - 1) / block_size.x,
        (pdm_shape[1] + block_size.y - 1) / block_size.y,
        (pdm_shape[2] + block_size.z - 1) / block_size.z
    );
    kernel_DryDistribution<<<block_number, block_size>>>(x.devptr, y.devptr, z.devptr, phi.devptr, aps.devptr, euler_eps, pdm_shape, gc);
}

void AlmDevCall::CalcTorqueAndThrust() {
    int n_ap_per_turbine = n_ap_per_blade*turbines.n_blade;
    for (int tid = 0; tid < turbines.n_turbine; tid ++) {
        thrust::device_ptr<Real> torque_ptr = thrust::device_pointer_cast(aps.dev.torque + n_ap_per_turbine*tid);
        thrust::device_ptr<Real> thrust_ptr = thrust::device_pointer_cast(aps.dev.thrust + n_ap_per_turbine*tid);
        turbines.host.torque[tid] = thrust::reduce(torque_ptr, torque_ptr + n_ap_per_turbine);
        turbines.host.thrust[tid] = thrust::reduce(thrust_ptr, thrust_ptr + n_ap_per_turbine);
    }
}

}

}