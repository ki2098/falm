#include "hip/hip_runtime.h"
#include "../typedef.h"
#include "../CPML1v2.h"
#include "devutil.cuh"

namespace Falm {

__global__ void kernel_CPM_PackBuffer(REAL *buffer, INTx3 buf_shape, INTx3 buf_offset, REAL *src, INTx3 src_shape) {
    INT i, j, k;
    GLOBAL_THREAD_IDX_3D(i, j, k);
    if (i < buf_shape.x && j < buf_shape.y && k < buf_shape.z) {
        INT buf_idx = IDX(i, j, k, buf_shape);
        i += buf_offset.x;
        j += buf_offset.y;
        k += buf_offset.z;
        INT src_idx = IDX(i, j, k, src_shape);
        buffer[buf_idx] = src[src_idx];
    }
}

__global__ void kernel_CPM_PackColoredBuffer(REAL *buffer, INTx3 buf_shape, INTx3 buf_offset, INT color, REAL *src, INTx3 src_shape, INTx3 src_offset) {
    INT i, j, k;
    GLOBAL_THREAD_IDX_3D(i, j, k);
    if (i < buf_shape.x && j < buf_shape.y && k < buf_shape.z) {
        INT buf_idx = IDX(i, j, k, buf_shape);
        i += buf_offset.x;
        j += buf_offset.y;
        k += buf_offset.z;
        INT src_idx = IDX(i, j, k, src_shape);
        if ((i + j + k + SUM3(src_offset)) % 2 == color) {
            buffer[buf_idx / 2] = src[src_idx];
        }
    }
}

__global__ void kernel_CPM_UnpackBuffer(REAL *buffer, INTx3 buf_shape, INTx3 buf_offset, REAL *dst, INTx3 dst_shape) {
    INT i, j, k;
    GLOBAL_THREAD_IDX_3D(i, j, k);
    if (i < buf_shape.x && j < buf_shape.y && k < buf_shape.z) {
        INT buf_idx = IDX(i, j, k, buf_shape);
        i += buf_offset.x;
        j += buf_offset.y;
        k += buf_offset.z;
        INT dst_idx = IDX(i, j, k, dst_shape);
        dst[dst_idx] = buffer[buf_idx];
    }
}

__global__ void kernel_CPM_UnpackColoredBuffer(REAL *buffer, INTx3 buf_shape, INTx3 buf_offset, INT color , REAL *dst, INTx3 dst_shape, INTx3 dst_offset) {
    INT i, j, k;
    GLOBAL_THREAD_IDX_3D(i, j, k);
    if (i < buf_shape.x && j < buf_shape.y && k < buf_shape.z) {
        INT buf_idx = IDX(i, j, k, buf_shape);
        i += buf_offset.x;
        j += buf_offset.y;
        k += buf_offset.z;
        INT dst_idx = IDX(i, j, k, dst_shape);
        if ((i + j + k + SUM3(dst_offset)) % 2 == color) {
            dst[dst_idx] = buffer[buf_idx / 2];
        }
    }
}

// void CPML1Dev_PackBuffer(CPMBuffer &buffer, REAL *src, Mapper &pdom, dim3 block_dim) {
//     Mapper &map = buffer.map;
//     dim3 grid_dim(
//         (map.shape.x + block_dim.x - 1) / block_dim.x,
//         (map.shape.y + block_dim.y - 1) / block_dim.y,
//         (map.shape.z + block_dim.z - 1) / block_dim.z
//     );
//     if (buffer.hdctype == HDCType::Device) {
//         kernel_CPM_PackBuffer<<<grid_dim, block_dim, 0, 0>>>((REAL*)buffer.ptr, buffer.map.shape, buffer.map.offset, src, pdom.shape);
//     } else if (buffer.hdctype == HDCType::Host) {
//         REAL *ptr = (REAL*)falmDevMalloc(sizeof(REAL) * buffer.count);
//         kernel_CPM_PackBuffer<<<grid_dim, block_dim, 0, 0>>>(ptr, buffer.map.shape, buffer.map.offset, src, pdom.shape);
//         falmMemcpy(buffer.ptr, ptr, sizeof(REAL) * buffer.count, MCpType::Dev2Hst);
//         falmDevFreePtr(ptr);
//     }
// }

// void CPML1Dev_PackColoredBuffer(CPMBuffer &buffer, REAL *src, Mapper &pdom, dim3 block_dim) {
//     Mapper &map = buffer.map;
//     dim3 grid_dim(
//         (map.shape.x + block_dim.x - 1) / block_dim.x,
//         (map.shape.y + block_dim.y - 1) / block_dim.y,
//         (map.shape.z + block_dim.z - 1) / block_dim.z
//     );
//     if (buffer.hdctype == HDCType::Device) {
//         kernel_CPM_PackColoredBuffer<<<grid_dim, block_dim, 0, 0>>>((REAL*)buffer.ptr, buffer.map.shape, buffer.map.offset, buffer.color, src, pdom.shape, pdom.offset);
//     } else if (buffer.hdctype == HDCType::Host) {
//         REAL *ptr = (REAL*)falmDevMalloc(sizeof(REAL) * buffer.count);
//         kernel_CPM_PackColoredBuffer<<<grid_dim, block_dim, 0, 0>>>(ptr, buffer.map.shape, buffer.map.offset, buffer.color, src, pdom.shape, pdom.offset);
//         falmMemcpy(buffer.ptr, ptr, sizeof(REAL) * buffer.count, MCpType::Dev2Hst);
//         falmDevFreePtr(ptr);
//     }
// }

// void CPML1Dev_UnpackBuffer(CPMBuffer &buffer, REAL *dst, Mapper &pdom, dim3 block_dim) {
//     Mapper &map = buffer.map;
//     dim3 grid_dim(
//         (map.shape.x + block_dim.x - 1) / block_dim.x,
//         (map.shape.y + block_dim.y - 1) / block_dim.y,
//         (map.shape.z + block_dim.z - 1) / block_dim.z
//     );
//     if (buffer.hdctype == HDCType::Device) {
//         kernel_CPM_UnpackBuffer<<<grid_dim, block_dim, 0, 0>>>((REAL*)buffer.ptr, buffer.map.shape, buffer.map.offset, dst, pdom.shape);
//     } else if (buffer.hdctype == HDCType::Host) {
//         REAL *ptr = (REAL*)falmDevMalloc(sizeof(REAL) * buffer.count);
//         falmMemcpy(ptr, buffer.ptr, sizeof(REAL) * buffer.count, MCpType::Hst2Dev);
//         kernel_CPM_UnpackBuffer<<<grid_dim, block_dim, 0, 0>>>(ptr, buffer.map.shape, buffer.map.offset, dst, pdom.shape);
//         falmDevFreePtr(ptr);
//     }
// }

// void CPML1Dev_UnpackColoredBuffer(CPMBuffer &buffer, REAL *dst, Mapper &pdom, dim3 block_dim) {
//     Mapper &map = buffer.map;
//     dim3 grid_dim(
//         (map.shape.x + block_dim.x - 1) / block_dim.x,
//         (map.shape.y + block_dim.y - 1) / block_dim.y,
//         (map.shape.z + block_dim.z - 1) / block_dim.z
//     );
//     if (buffer.hdctype == HDCType::Device) {
//         kernel_CPM_UnpackColoredBuffer<<<grid_dim, block_dim, 0, 0>>>((REAL*)buffer.ptr, buffer.map.shape, buffer.map.offset, buffer.color, dst, pdom.shape, pdom.offset);
//     } else if (buffer.hdctype == HDCType::Host) {
//         REAL *ptr = (REAL*)falmDevMalloc(sizeof(REAL) * buffer.count);
//         falmMemcpy(ptr, buffer.ptr, sizeof(REAL) * buffer.count, MCpType::Hst2Dev);
//         kernel_CPM_UnpackColoredBuffer<<<grid_dim, block_dim, 0, 0>>>(ptr, buffer.map.shape, buffer.map.offset, buffer.color, dst, pdom.shape, pdom.offset);
//         falmDevFreePtr(ptr);
//     }
// }

void CPML0Dev_PackBuffer(REAL *buffer, Mapper &map, REAL *src, Mapper &proc, dim3 block_dim, STREAM stream) {
    dim3 grid_dim(
        (map.shape.x + block_dim.x - 1) / block_dim.x,
        (map.shape.y + block_dim.y - 1) / block_dim.y,
        (map.shape.z + block_dim.z - 1) / block_dim.z
    );
    kernel_CPM_PackBuffer<<<grid_dim, block_dim, 0, stream>>>(buffer, map.shape, map.offset, src, proc.shape);
}

void CPML0Dev_PackColoredBuffer(REAL *buffer, Mapper &map, INT color, REAL *src, Mapper &proc, dim3 block_dim, STREAM stream) {
    dim3 grid_dim(
        (map.shape.x + block_dim.x - 1) / block_dim.x,
        (map.shape.y + block_dim.y - 1) / block_dim.y,
        (map.shape.z + block_dim.z - 1) / block_dim.z
    );
    kernel_CPM_PackColoredBuffer<<<grid_dim, block_dim, 0, stream>>>(buffer, map.shape, map.offset, color, src, proc.shape, proc.offset);
}

void CPML0Dev_UnpackBuffer(REAL *buffer, Mapper &map, REAL *dst, Mapper &proc, dim3 block_dim, STREAM stream) {
    dim3 grid_dim(
        (map.shape.x + block_dim.x - 1) / block_dim.x,
        (map.shape.y + block_dim.y - 1) / block_dim.y,
        (map.shape.z + block_dim.z - 1) / block_dim.z
    );
    kernel_CPM_UnpackBuffer<<<grid_dim, block_dim, 0, stream>>>(buffer, map.shape, map.offset, dst, proc.shape);
}

void CPML0Dev_UnpackColoredBuffer(REAL *buffer, Mapper &map, INT color, REAL *dst, Mapper &proc, dim3 block_dim, STREAM stream) {
    dim3 grid_dim(
        (map.shape.x + block_dim.x - 1) / block_dim.x,
        (map.shape.y + block_dim.y - 1) / block_dim.y,
        (map.shape.z + block_dim.z - 1) / block_dim.z
    );
    kernel_CPM_UnpackColoredBuffer<<<grid_dim, block_dim, 0, stream>>>(buffer, map.shape, map.offset, color, dst, proc.shape, proc.offset);
}

}
