#include "hip/hip_runtime.h"
#include "../structEqL1.h"
#include "../MVL1.h"
#include "devutil.cuh"

namespace Falm {

__global__ void kernel_Struct3d7p_MV(const MatrixFrame<REAL> *va, const MatrixFrame<REAL> *vx, const MatrixFrame<REAL> *vax, INTx3 pdm_shape, INTx3 map_shape, INTx3 map_offset) {
    const MatrixFrame<REAL> &a=*va, &x=*vx, &ax=*vax;
    INT i, j, k;
    GLOBAL_THREAD_IDX_3D(i, j, k);
    if (i < map_shape.x && j < map_shape.y && k < map_shape.z) {
        i += map_offset.x;
        j += map_offset.y;
        k += map_offset.z;
        INT idxc = IDX(i  , j  , k  , pdm_shape);
        INT idxe = IDX(i+1, j  , k  , pdm_shape);
        INT idxw = IDX(i-1, j  , k  , pdm_shape);
        INT idxn = IDX(i  , j+1, k  , pdm_shape);
        INT idxs = IDX(i  , j-1, k  , pdm_shape);
        INT idxt = IDX(i  , j  , k+1, pdm_shape);
        INT idxb = IDX(i  , j  , k-1, pdm_shape);
        REAL ac = a(idxc, 0);
        REAL ae = a(idxc, 1);
        REAL aw = a(idxc, 2);
        REAL an = a(idxc, 3);
        REAL as = a(idxc, 4);
        REAL at = a(idxc, 5);
        REAL ab = a(idxc, 6);
        REAL xc = x(idxc);
        REAL xe = x(idxe);
        REAL xw = x(idxw);
        REAL xn = x(idxn);
        REAL xs = x(idxs);
        REAL xt = x(idxt);
        REAL xb = x(idxb);
        ax(idxc) = ac * xc + ae * xe + aw * xw + an * xn + as * xs + at * xt + ab * xb;
    }
}

void L0Dev_Struct3d7p_MV(Matrix<REAL> &a, Matrix<REAL> &x, Matrix<REAL> &ax, Region &pdm, const Region &map, dim3 block_dim, STREAM stream) {
    assert(
        a.shape.x == x.shape.x && a.shape.x == ax.shape.x &&
        a.shape.y == 7 && x.shape.y == 1 && ax.shape.y == 1
    );
    dim3 grid_dim(
        (map.shape.x + block_dim.x - 1) / block_dim.x,
        (map.shape.y + block_dim.y - 1) / block_dim.y,
        (map.shape.z + block_dim.z - 1) / block_dim.z
    );

    kernel_Struct3d7p_MV<<<grid_dim, block_dim, 0, stream>>>(a.devptr, x.devptr, ax.devptr, pdm.shape, map.shape, map.offset);
}

__global__ void kernel_Struct3d7p_Res(const MatrixFrame<REAL> *va, const MatrixFrame<REAL> *vx, const MatrixFrame<REAL> *vb, const MatrixFrame<REAL> *vr, INTx3 pdm_shape, INTx3 map_shape, INTx3 map_offset) {
    const MatrixFrame<REAL> &a=*va, &x=*vx, &b=*vb, &r=*vr;
    INT i, j, k;
    GLOBAL_THREAD_IDX_3D(i, j, k);
    if (i < map_shape.x && j < map_shape.y && k < map_shape.z) {
        i += map_offset.x;
        j += map_offset.y;
        k += map_offset.z;
        INT idxc = IDX(i  , j  , k  , pdm_shape);
        INT idxe = IDX(i+1, j  , k  , pdm_shape);
        INT idxw = IDX(i-1, j  , k  , pdm_shape);
        INT idxn = IDX(i  , j+1, k  , pdm_shape);
        INT idxs = IDX(i  , j-1, k  , pdm_shape);
        INT idxt = IDX(i  , j  , k+1, pdm_shape);
        INT idxb = IDX(i  , j  , k-1, pdm_shape);
        REAL ac = a(idxc, 0);
        REAL ae = a(idxc, 1);
        REAL aw = a(idxc, 2);
        REAL an = a(idxc, 3);
        REAL as = a(idxc, 4);
        REAL at = a(idxc, 5);
        REAL ab = a(idxc, 6);
        REAL xc = x(idxc);
        REAL xe = x(idxe);
        REAL xw = x(idxw);
        REAL xn = x(idxn);
        REAL xs = x(idxs);
        REAL xt = x(idxt);
        REAL xb = x(idxb);
        r(idxc) = b(idxc) - (ac * xc + ae * xe + aw * xw + an * xn + as * xs + at * xt + ab * xb);
    }
}

void L0Dev_Struct3d7p_Res(Matrix<REAL> &a, Matrix<REAL> &x, Matrix<REAL> &b, Matrix<REAL> &r, Region &pdm, const Region &map, dim3 block_dim, STREAM stream) {
    assert(
        a.shape.x == x.shape.x && a.shape.x == b.shape.x && a.shape.x == r.shape.x &&
        a.shape.y == 7 && x.shape.y == 1 && b.shape.y == 1 && r.shape.y == 1
    );
    dim3 grid_dim(
        (map.shape.x + block_dim.x - 1) / block_dim.x,
        (map.shape.y + block_dim.y - 1) / block_dim.y,
        (map.shape.z + block_dim.z - 1) / block_dim.z
    );

    kernel_Struct3d7p_Res<<<grid_dim, block_dim, 0, stream>>>(a.devptr, x.devptr, b.devptr, r.devptr, pdm.shape, map.shape, map.offset);
}

__global__ void kernel_Struct3d7p_Jacobi(const MatrixFrame<REAL> *va, const MatrixFrame<REAL> *vx, const MatrixFrame<REAL> *vxp, const MatrixFrame<REAL> *vb, INTx3 pdm_shape, INTx3 map_shape, INTx3 map_offset) {
    const MatrixFrame<REAL> &a=*va, &x=*vx, &xp=*vxp, &b=*vb;
    INT i, j, k;
    GLOBAL_THREAD_IDX_3D(i, j, k);
    if (i < map_shape.x && j < map_shape.y && k < map_shape.z) {
        i += map_offset.x;
        j += map_offset.y;
        k += map_offset.z;
        INT idxc = IDX(i  , j  , k  , pdm_shape);
        INT idxe = IDX(i+1, j  , k  , pdm_shape);
        INT idxw = IDX(i-1, j  , k  , pdm_shape);
        INT idxn = IDX(i  , j+1, k  , pdm_shape);
        INT idxs = IDX(i  , j-1, k  , pdm_shape);
        INT idxt = IDX(i  , j  , k+1, pdm_shape);
        INT idxb = IDX(i  , j  , k-1, pdm_shape);
        REAL ac =  a(idxc, 0);
        REAL ae =  a(idxc, 1);
        REAL aw =  a(idxc, 2);
        REAL an =  a(idxc, 3);
        REAL as =  a(idxc, 4);
        REAL at =  a(idxc, 5);
        REAL ab =  a(idxc, 6);
        REAL xc = xp(idxc);
        REAL xe = xp(idxe);
        REAL xw = xp(idxw);
        REAL xn = xp(idxn);
        REAL xs = xp(idxs);
        REAL xt = xp(idxt);
        REAL xb = xp(idxb);
        x(idxc) = xc + (b(idxc) - (ac * xc + ae * xe + aw * xw + an * xn + as * xs + at * xt + ab * xb)) / ac;
    }
}

void L1EqSolver::L0Dev_Struct3d7p_JacobiSweep(Matrix<REAL> &a, Matrix<REAL> &x, Matrix<REAL> &xp, Matrix<REAL> &b, Region &pdm, const Region &map, dim3 block_dim, STREAM stream) {
    dim3 grid_dim(
        (map.shape.x + block_dim.x - 1) / block_dim.x,
        (map.shape.y + block_dim.y - 1) / block_dim.y,
        (map.shape.z + block_dim.z - 1) / block_dim.z
    );

    kernel_Struct3d7p_Jacobi<<<grid_dim, block_dim, 0, stream>>>(a.devptr, x.devptr, xp.devptr, b.devptr, pdm.shape, map.shape, map.offset);
}

void L1EqSolver::L1Dev_Struct3d7p_Jacobi(Matrix<REAL> &a, Matrix<REAL> &x, Matrix<REAL> &b, Matrix<REAL> &r, Region &pdm, INT gc, dim3 block_dim) {
    Region map(pdm.shape, gc);
    assert(
        a.shape.x == x.shape.x && a.shape.x == b.shape.x && a.shape.x == r.shape.x &&
        a.shape.y == 7 && x.shape.y == 1 && b.shape.y == 1 && r.shape.y == 1
    );

    Matrix<REAL> xp(x.shape.x, x.shape.y, HDCType::Device, "Jacobi" + x.name + "Previous");
    it = 0;
    do {
        xp.cpy(x, HDCType::Device);
        L0Dev_Struct3d7p_JacobiSweep(a, x, xp, b, pdm, map, block_dim);
        L0Dev_Struct3d7p_Res(a, x, b, r, pdm, map, block_dim);
        err = sqrt(L0Dev_EuclideanNormSq(r, pdm, map, block_dim)) / map.size;
        it ++;
    } while (it < maxit && err > tol);
}

void L1EqSolver::L1Dev_Struct3d7p_JacobiPC(Matrix<REAL> &a, Matrix<REAL> &x, Matrix<REAL> &b, Region &pdm, INT gc, dim3 block_dim) {
    Region map(pdm.shape, gc);
    Matrix<REAL> xp(x.shape.x, x.shape.y, HDCType::Device, "Jacobi" + x.name + "Previous");
    INT __it = 0;
    do {
        xp.cpy(x, HDCType::Device);
        L0Dev_Struct3d7p_JacobiSweep(a, x, xp, b, pdm, map, block_dim);
        __it ++;
    } while (__it < pc_maxit);
}

__global__ void kernel_Struct3d7p_SOR(const MatrixFrame<REAL> *va, const MatrixFrame<REAL> *vx, const MatrixFrame<REAL> *vb, REAL omega, INT color, INTx3 pdm_shape, INTx3 pdm_offset, INTx3 map_shape, INTx3 map_offset) {
    const MatrixFrame<REAL> &a=*va, &x=*vx, &b=*vb;
    INT i, j, k;
    GLOBAL_THREAD_IDX_3D(i, j, k);
    if (i < map_shape.x && j < map_shape.y && k < map_shape.z) {
        i += map_offset.x;
        j += map_offset.y;
        k += map_offset.z;
        INT idxc = IDX(i  , j  , k  , pdm_shape);
        INT idxe = IDX(i+1, j  , k  , pdm_shape);
        INT idxw = IDX(i-1, j  , k  , pdm_shape);
        INT idxn = IDX(i  , j+1, k  , pdm_shape);
        INT idxs = IDX(i  , j-1, k  , pdm_shape);
        INT idxt = IDX(i  , j  , k+1, pdm_shape);
        INT idxb = IDX(i  , j  , k-1, pdm_shape);
        REAL ac = a(idxc, 0);
        REAL ae = a(idxc, 1);
        REAL aw = a(idxc, 2);
        REAL an = a(idxc, 3);
        REAL as = a(idxc, 4);
        REAL at = a(idxc, 5);
        REAL ab = a(idxc, 6);
        REAL xc = x(idxc);
        REAL xe = x(idxe);
        REAL xw = x(idxw);
        REAL xn = x(idxn);
        REAL xs = x(idxs);
        REAL xt = x(idxt);
        REAL xb = x(idxb);
        REAL bc = b(idxc);
        REAL cc = 0;
        if ((i + j + k + SUM3(pdm_offset)) % 2 == color) {
            cc = (bc - (ac * xc + ae * xe + aw * xw + an * xn + as * xs + at * xt + ab * xb)) / ac;
        }
        x(idxc) = xc + omega * cc;
    }
}

void L1EqSolver::L0Dev_Struct3d7p_SORSweep(Matrix<REAL> &a, Matrix<REAL> &x, Matrix<REAL> &b, REAL omega, INT color, Region &pdm, const Region &map, dim3 block_dim, STREAM stream) {
    dim3 grid_dim(
        (map.shape.x + block_dim.x - 1) / block_dim.x,
        (map.shape.y + block_dim.y - 1) / block_dim.y,
        (map.shape.z + block_dim.z - 1) / block_dim.z
    );

    kernel_Struct3d7p_SOR<<<grid_dim, block_dim, 0, stream>>>(a.devptr, x.devptr, b.devptr, omega, color, pdm.shape, pdm.offset, map.shape, map.offset);
}

void L1EqSolver::L1Dev_Struct3d7p_SOR(Matrix<REAL> &a, Matrix<REAL> &x, Matrix<REAL> &b, Matrix<REAL> &r, Region &pdm, INT gc, dim3 block_dim) {
    Region map(pdm.shape, gc);
    assert(
        a.shape.x == x.shape.x && a.shape.x == b.shape.x && a.shape.x == r.shape.x &&
        a.shape.y == 7 && x.shape.y == 1 && b.shape.y == 1 && r.shape.y == 1
    );

    it = 0;
    do {
        L0Dev_Struct3d7p_SORSweep(a, x, b, relax_factor, Color::Black, pdm, map, block_dim);
        L0Dev_Struct3d7p_SORSweep(a, x, b, relax_factor, Color::Red  , pdm, map, block_dim);
        L0Dev_Struct3d7p_Res(a, x, b, r, pdm, map, block_dim);
        err = sqrt(L0Dev_EuclideanNormSq(r, pdm, map, block_dim)) / map.size;
        it ++;
    } while (it < maxit && err > tol);
}

void L1EqSolver::L1Dev_Struct3d7p_SORPC(Matrix<REAL> &a, Matrix<REAL> &x, Matrix<REAL> &b, Region &pdm, INT gc, dim3 block_dim) {
    Region map(pdm.shape, gc);
    INT __it = 0;
    do {
        L0Dev_Struct3d7p_SORSweep(a, x, b, pc_relax_factor, Color::Black, pdm, map, block_dim);
        L0Dev_Struct3d7p_SORSweep(a, x, b, pc_relax_factor, Color::Red  , pdm, map, block_dim);
         __it ++;
    } while (__it < pc_maxit);
}

__global__ void kernel_PBiCGStab_1(const MatrixFrame<REAL> *vp, const MatrixFrame<REAL> *vq, const MatrixFrame<REAL> *vr, REAL beta, REAL omega, INTx3 pdm_shape, INTx3 map_shape, INTx3 map_offset) {
    const MatrixFrame<REAL> &p=*vp, &q=*vq, &r=*vr;
    INT i, j, k;
    GLOBAL_THREAD_IDX_3D(i, j, k);
    if (i < map_shape.x && j < map_shape.y && k < map_shape.z) {
        i += map_offset.x;
        j += map_offset.y;
        k += map_offset.z;
        INT idx = IDX(i, j, k, pdm_shape);
        p(idx) = r(idx) + beta * (p(idx) - omega * q(idx));
    }
}

void L1EqSolver::L0Dev_PBiCGStab1(Matrix<REAL> &p, Matrix<REAL> &q, Matrix<REAL> &r, REAL beta, REAL omega, Region &pdm, const Region &map, dim3 block_dim) {
    dim3 grid_dim(
        (map.shape.x + block_dim.x - 1) / block_dim.x,
        (map.shape.y + block_dim.y - 1) / block_dim.y,
        (map.shape.z + block_dim.z - 1) / block_dim.z
    );
    kernel_PBiCGStab_1<<<grid_dim, block_dim, 0, 0>>>(p.devptr, q.devptr, r.devptr, beta, omega, pdm.shape, map.shape, map.offset);
}

__global__ void kernel_PBiCGStab_2(const MatrixFrame<REAL> *vs, const MatrixFrame<REAL> *vq, const MatrixFrame<REAL> *vr, REAL alpha, INTx3 pdm_shape, INTx3 map_shape, INTx3 map_offset) {
    const MatrixFrame<REAL> &s=*vs, &q=*vq, &r=*vr;
    INT i, j, k;
    GLOBAL_THREAD_IDX_3D(i, j, k);
    if (i < map_shape.x && j < map_shape.y && k < map_shape.z) {
        i += map_offset.x;
        j += map_offset.y;
        k += map_offset.z;
        INT idx = IDX(i, j, k, pdm_shape);
        s(idx) = r(idx) - alpha * q(idx);
    }
}

void L1EqSolver::L0Dev_PBiCGStab2(Matrix<REAL> &s, Matrix<REAL> &q, Matrix<REAL> &r, REAL alpha, Region &pdm, const Region &map, dim3 block_dim) {
    dim3 grid_dim(
        (map.shape.x + block_dim.x - 1) / block_dim.x,
        (map.shape.y + block_dim.y - 1) / block_dim.y,
        (map.shape.z + block_dim.z - 1) / block_dim.z
    );
    kernel_PBiCGStab_2<<<grid_dim, block_dim, 0, 0>>>(s.devptr, q.devptr, r.devptr, alpha, pdm.shape, map.shape, map.offset);
}

__global__ void kernel_PBiCGStab_3(const MatrixFrame<REAL> *vx, const MatrixFrame<REAL> *vpp, const MatrixFrame<REAL> *vss, REAL alpha, REAL omega, INTx3 pdm_shape, INTx3 map_shape, INTx3 map_offset) {
    const MatrixFrame<REAL> &x=*vx, &pp=*vpp, &ss=*vss;
    INT i, j, k;
    GLOBAL_THREAD_IDX_3D(i, j, k);
    if (i < map_shape.x && j < map_shape.y && k < map_shape.z) {
        i += map_offset.x;
        j += map_offset.y;
        k += map_offset.z;
        INT idx = IDX(i, j, k, pdm_shape);
        x(idx) += alpha * pp(idx) + omega * ss(idx);
    }
}

void L1EqSolver::L0Dev_PBiCGStab3(Matrix<REAL> &x, Matrix<REAL> &pp, Matrix<REAL> &ss, REAL alpha, REAL omega, Region &pdm, const Region &map, dim3 block_dim) {
    dim3 grid_dim(
        (map.shape.x + block_dim.x - 1) / block_dim.x,
        (map.shape.y + block_dim.y - 1) / block_dim.y,
        (map.shape.z + block_dim.z - 1) / block_dim.z
    );
    kernel_PBiCGStab_3<<<grid_dim, block_dim, 0, 0>>>(x.devptr, pp.devptr, ss.devptr, alpha, omega, pdm.shape, map.shape, map.offset);
} 

__global__ void kernel_PBiCGStab_4(const MatrixFrame<REAL> *vr, const MatrixFrame<REAL> *vs, const MatrixFrame<REAL> *vt, REAL omega, INTx3 pdm_shape, INTx3 map_shape, INTx3 map_offset) {
    const MatrixFrame<REAL> &r=*vr, &s=*vs, &t=*vt;
    INT i, j, k;
    GLOBAL_THREAD_IDX_3D(i, j, k);
    if (i < map_shape.x && j < map_shape.y && k < map_shape.z) {
        i += map_offset.x;
        j += map_offset.y;
        k += map_offset.z;
        INT idx = IDX(i, j, k, pdm_shape);
        r(idx) = s(idx) - omega * t(idx);
    }
}

void L1EqSolver::L0Dev_PBiCGStab4(Matrix<REAL> &r, Matrix<REAL> &s, Matrix<REAL> &t, REAL omega, Region &pdm, const Region &map, dim3 block_dim) {
    dim3 grid_dim(
        (map.shape.x + block_dim.x - 1) / block_dim.x,
        (map.shape.y + block_dim.y - 1) / block_dim.y,
        (map.shape.z + block_dim.z - 1) / block_dim.z
    );
    kernel_PBiCGStab_4<<<grid_dim, block_dim, 0, 0>>>(r.devptr, s.devptr, t.devptr, omega, pdm.shape, map.shape, map.offset);
}

void L1EqSolver::L1Dev_Struct3d7p_PBiCGStab(Matrix<REAL> &a, Matrix<REAL> &x, Matrix<REAL> &b, Matrix<REAL> &r, Region &pdm, INT gc, dim3 block_dim) {
    Region map(pdm.shape, gc);
    assert(
        a.shape.x == x.shape.x && a.shape.x == b.shape.x && a.shape.x == r.shape.x &&
        a.shape.y == 7 && x.shape.y == 1 && b.shape.y == 1 && r.shape.y == 1
    );
    dim3 grid_dim(
        (map.shape.x + block_dim.x - 1) / block_dim.x,
        (map.shape.y + block_dim.y - 1) / block_dim.y,
        (map.shape.z + block_dim.z - 1) / block_dim.z
    );

    Matrix<REAL> rr(pdm.shape, 1, HDCType::Device, "PBiCGStab rr");
    Matrix<REAL>  p(pdm.shape, 1, HDCType::Device, "PBiCGStab  p");
    Matrix<REAL>  q(pdm.shape, 1, HDCType::Device, "PBiCGStab  q");
    Matrix<REAL>  s(pdm.shape, 1, HDCType::Device, "PBiCGStab  s");
    Matrix<REAL> pp(pdm.shape, 1, HDCType::Device, "PBiCGStab pp");
    Matrix<REAL> ss(pdm.shape, 1, HDCType::Device, "PBiCGStab ss");
    Matrix<REAL>  t(pdm.shape, 1, HDCType::Device, "PBiCGStab  t");

    REAL rho, rrho, alpha, beta, omega;

    L0Dev_Struct3d7p_Res(a, x, b, r, pdm, map, block_dim);
    err = sqrt(L0Dev_EuclideanNormSq(r, pdm, map, block_dim)) / map.size;
    rr.cpy(r, HDCType::Device);

    rrho  = 1.0;
    alpha = 0.0;
    omega = 1.0;

    it = 0;
    do {
        // if (err < tol) {
        //     break;
        // }

        rho = L0Dev_DotProduct(r, rr, pdm, map, block_dim);
        if (fabs(rho) < __FLT_MIN__) {
            err = rho;
            break;
        }

        if (it == 0) {
            p.cpy(r, HDCType::Device);
        } else {
            beta = (rho * alpha) / (rrho * omega);
            // kernel_PBiCGStab_1<<<grid_dim, block_dim>>>(p.devptr, q.devptr, r.devptr, beta, omega, pdm.shape, map.shape, map.offset);
            L0Dev_PBiCGStab1(p, q, r, beta, omega, pdm, map, block_dim);
        }
        pp.clear(HDCType::Device);
        L1Dev_Struct3d7p_Precondition(a, pp, p, pdm, gc, block_dim);
        L0Dev_Struct3d7p_MV(a, pp, q, pdm, map, block_dim);
        alpha = rho / L0Dev_DotProduct(rr, q, pdm, map, block_dim);

        // kernel_PBiCGStab_2<<<grid_dim, block_dim>>>(s.devptr, q.devptr, r.devptr, alpha, pdm.shape, map.shape, map.offset);
        L0Dev_PBiCGStab2(s, q, r, alpha, pdm, map, block_dim);
        ss.clear(HDCType::Device);
        L1Dev_Struct3d7p_Precondition(a, ss, s, pdm, gc, block_dim);
        L0Dev_Struct3d7p_MV(a, ss, t, pdm, map, block_dim);
        omega = L0Dev_DotProduct(t, s, pdm, map, block_dim) / L0Dev_DotProduct(t, t, pdm, map, block_dim);

        // kernel_PBiCGStab_3<<<grid_dim, block_dim, 0, 0>>>(x.devptr, pp.devptr, ss.devptr, alpha, omega, pdm.shape, map.shape, map.offset);
        // kernel_PBiCGStab_4<<<grid_dim, block_dim, 0, 0>>>(r.devptr, s.devptr, t.devptr, omega, pdm.shape, map.shape, map.offset);
        L0Dev_PBiCGStab3(x, pp, ss, alpha, omega, pdm, map, block_dim);
        L0Dev_PBiCGStab4(r, s, t, omega, pdm, map, block_dim);

        rrho = rho;

        err = sqrt(L0Dev_EuclideanNormSq(r, pdm, map, block_dim)) / map.size;
        it ++;
    } while (it < maxit && err > tol);
}

}
