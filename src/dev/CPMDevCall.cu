#include "hip/hip_runtime.h"
#include "../typedef.h"
#include "../CPMDevCall.h"
#include "devutil.cuh"

namespace Falm {

__global__ void kernel_CPM_PackBuffer(REAL *buffer, INT3 buf_shape, INT3 buf_offset, REAL *src, INT3 src_shape) {
    INT i, j, k;
    GLOBAL_THREAD_IDX_3D(i, j, k);
    if (i < buf_shape[0] && j < buf_shape[1] && k < buf_shape[2]) {
        INT buf_idx = IDX(i, j, k, buf_shape);
        i += buf_offset[0];
        j += buf_offset[1];
        k += buf_offset[2];
        INT src_idx = IDX(i, j, k, src_shape);
        buffer[buf_idx] = src[src_idx];
    }
}

__global__ void kernel_CPM_PackColoredBuffer(REAL *buffer, INT3 buf_shape, INT3 buf_offset, INT color, REAL *src, INT3 src_shape, INT3 src_offset) {
    INT i, j, k;
    GLOBAL_THREAD_IDX_3D(i, j, k);
    if (i < buf_shape[0] && j < buf_shape[1] && k < buf_shape[2]) {
        INT buf_idx = IDX(i, j, k, buf_shape);
        i += buf_offset[0];
        j += buf_offset[1];
        k += buf_offset[2];
        INT src_idx = IDX(i, j, k, src_shape);
        if ((i + j + k + SUM3(src_offset)) % 2 == color) {
            buffer[buf_idx / 2] = src[src_idx];
        }
    }
}

__global__ void kernel_CPM_UnpackBuffer(REAL *buffer, INT3 buf_shape, INT3 buf_offset, REAL *dst, INT3 dst_shape) {
    INT i, j, k;
    GLOBAL_THREAD_IDX_3D(i, j, k);
    if (i < buf_shape[0] && j < buf_shape[1] && k < buf_shape[2]) {
        INT buf_idx = IDX(i, j, k, buf_shape);
        i += buf_offset[0];
        j += buf_offset[1];
        k += buf_offset[2];
        INT dst_idx = IDX(i, j, k, dst_shape);
        dst[dst_idx] = buffer[buf_idx];
    }
}

__global__ void kernel_CPM_UnpackColoredBuffer(REAL *buffer, INT3 buf_shape, INT3 buf_offset, INT color , REAL *dst, INT3 dst_shape, INT3 dst_offset) {
    INT i, j, k;
    GLOBAL_THREAD_IDX_3D(i, j, k);
    if (i < buf_shape[0] && j < buf_shape[1] && k < buf_shape[2]) {
        INT buf_idx = IDX(i, j, k, buf_shape);
        i += buf_offset[0];
        j += buf_offset[1];
        k += buf_offset[2];
        INT dst_idx = IDX(i, j, k, dst_shape);
        if ((i + j + k + SUM3(dst_offset)) % 2 == color) {
            dst[dst_idx] = buffer[buf_idx / 2];
        }
    }
}

void CPMDevCall::PackBuffer(REAL *buffer, Region &map, REAL *src, Region &pdm, dim3 block_dim, STREAM stream) {
    dim3 grid_dim(
        (map.shape[0] + block_dim.x - 1) / block_dim.x,
        (map.shape[1] + block_dim.y - 1) / block_dim.y,
        (map.shape[2] + block_dim.z - 1) / block_dim.z
    );
    kernel_CPM_PackBuffer<<<grid_dim, block_dim, 0, stream>>>(buffer, map.shape, map.offset, src, pdm.shape);
}

void CPMDevCall::PackColoredBuffer(REAL *buffer, Region &map, INT color, REAL *src, Region &pdm, dim3 block_dim, STREAM stream) {
    dim3 grid_dim(
        (map.shape[0] + block_dim.x - 1) / block_dim.x,
        (map.shape[1] + block_dim.y - 1) / block_dim.y,
        (map.shape[2] + block_dim.z - 1) / block_dim.z
    );
    kernel_CPM_PackColoredBuffer<<<grid_dim, block_dim, 0, stream>>>(buffer, map.shape, map.offset, color, src, pdm.shape, pdm.offset);
}

void CPMDevCall::UnpackBuffer(REAL *buffer, Region &map, REAL *dst, Region &pdm, dim3 block_dim, STREAM stream) {
    dim3 grid_dim(
        (map.shape[0] + block_dim.x - 1) / block_dim.x,
        (map.shape[1] + block_dim.y - 1) / block_dim.y,
        (map.shape[2] + block_dim.z - 1) / block_dim.z
    );
    kernel_CPM_UnpackBuffer<<<grid_dim, block_dim, 0, stream>>>(buffer, map.shape, map.offset, dst, pdm.shape);
}

void CPMDevCall::UnpackColoredBuffer(REAL *buffer, Region &map, INT color, REAL *dst, Region &pdm, dim3 block_dim, STREAM stream) {
    dim3 grid_dim(
        (map.shape[0] + block_dim.x - 1) / block_dim.x,
        (map.shape[1] + block_dim.y - 1) / block_dim.y,
        (map.shape[2] + block_dim.z - 1) / block_dim.z
    );
    kernel_CPM_UnpackColoredBuffer<<<grid_dim, block_dim, 0, stream>>>(buffer, map.shape, map.offset, color, dst, pdm.shape, pdm.offset);
}

void CPML1Dev_PackBuffer(CPMBuffer &buffer, REAL *src, Region &pdm, dim3 block_dim) {
    Region &map = buffer.map;
    dim3 grid_dim(
        (map.shape[0] + block_dim.x - 1) / block_dim.x,
        (map.shape[1] + block_dim.y - 1) / block_dim.y,
        (map.shape[2] + block_dim.z - 1) / block_dim.z
    );
    if (buffer.hdctype == HDCType::Device) {
        kernel_CPM_PackBuffer<<<grid_dim, block_dim, 0, 0>>>((REAL*)buffer.ptr, buffer.map.shape, buffer.map.offset, src, pdm.shape);
    } else if (buffer.hdctype == HDCType::Host) {
        REAL *ptr = (REAL*)falmMallocDevice(sizeof(REAL) * buffer.count);
        kernel_CPM_PackBuffer<<<grid_dim, block_dim, 0, 0>>>(ptr, buffer.map.shape, buffer.map.offset, src, pdm.shape);
        falmMemcpy(buffer.ptr, ptr, sizeof(REAL) * buffer.count, MCpType::Dev2Hst);
        falmFreeDevice(ptr);
    }
}

void CPML1Dev_PackColoredBuffer(CPMBuffer &buffer, REAL *src, Region &pdm, dim3 block_dim) {
    Region &map = buffer.map;
    dim3 grid_dim(
        (map.shape[0] + block_dim.x - 1) / block_dim.x,
        (map.shape[1] + block_dim.y - 1) / block_dim.y,
        (map.shape[2] + block_dim.z - 1) / block_dim.z
    );
    if (buffer.hdctype == HDCType::Device) {
        kernel_CPM_PackColoredBuffer<<<grid_dim, block_dim, 0, 0>>>((REAL*)buffer.ptr, buffer.map.shape, buffer.map.offset, buffer.color, src, pdm.shape, pdm.offset);
    } else if (buffer.hdctype == HDCType::Host) {
        REAL *ptr = (REAL*)falmMallocDevice(sizeof(REAL) * buffer.count);
        kernel_CPM_PackColoredBuffer<<<grid_dim, block_dim, 0, 0>>>(ptr, buffer.map.shape, buffer.map.offset, buffer.color, src, pdm.shape, pdm.offset);
        falmMemcpy(buffer.ptr, ptr, sizeof(REAL) * buffer.count, MCpType::Dev2Hst);
        falmFreeDevice(ptr);
    }
}

void CPML1Dev_UnpackBuffer(CPMBuffer &buffer, REAL *dst, Region &pdm, dim3 block_dim) {
    Region &map = buffer.map;
    dim3 grid_dim(
        (map.shape[0] + block_dim.x - 1) / block_dim.x,
        (map.shape[1] + block_dim.y - 1) / block_dim.y,
        (map.shape[2] + block_dim.z - 1) / block_dim.z
    );
    if (buffer.hdctype == HDCType::Device) {
        kernel_CPM_UnpackBuffer<<<grid_dim, block_dim, 0, 0>>>((REAL*)buffer.ptr, buffer.map.shape, buffer.map.offset, dst, pdm.shape);
    } else if (buffer.hdctype == HDCType::Host) {
        REAL *ptr = (REAL*)falmMallocDevice(sizeof(REAL) * buffer.count);
        falmMemcpy(ptr, buffer.ptr, sizeof(REAL) * buffer.count, MCpType::Hst2Dev);
        kernel_CPM_UnpackBuffer<<<grid_dim, block_dim, 0, 0>>>(ptr, buffer.map.shape, buffer.map.offset, dst, pdm.shape);
        falmFreeDevice(ptr);
    }
}

void CPML1Dev_UnpackColoredBuffer(CPMBuffer &buffer, REAL *dst, Region &pdm, dim3 block_dim) {
    Region &map = buffer.map;
    dim3 grid_dim(
        (map.shape[0] + block_dim.x - 1) / block_dim.x,
        (map.shape[1] + block_dim.y - 1) / block_dim.y,
        (map.shape[2] + block_dim.z - 1) / block_dim.z
    );
    if (buffer.hdctype == HDCType::Device) {
        kernel_CPM_UnpackColoredBuffer<<<grid_dim, block_dim, 0, 0>>>((REAL*)buffer.ptr, buffer.map.shape, buffer.map.offset, buffer.color, dst, pdm.shape, pdm.offset);
    } else if (buffer.hdctype == HDCType::Host) {
        REAL *ptr = (REAL*)falmMallocDevice(sizeof(REAL) * buffer.count);
        falmMemcpy(ptr, buffer.ptr, sizeof(REAL) * buffer.count, MCpType::Hst2Dev);
        kernel_CPM_UnpackColoredBuffer<<<grid_dim, block_dim, 0, 0>>>(ptr, buffer.map.shape, buffer.map.offset, buffer.color, dst, pdm.shape, pdm.offset);
        falmFreeDevice(ptr);
    }
}

}
